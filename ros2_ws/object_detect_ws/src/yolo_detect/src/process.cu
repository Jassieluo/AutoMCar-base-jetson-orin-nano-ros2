#include "hip/hip_runtime.h"
#include "yolo_detect/process.h"


__global__ void warpaffine_kernel(
	uint8_t* src, int src_line_size, int src_width,
	int src_height, float* dst, int dst_width,
	int dst_height, uint8_t const_value_st,
	AffineMatrix d2s, int edge) {
	int position = blockDim.x * blockIdx.x + threadIdx.x;
	if (position >= edge) return;

	float m_x1 = d2s.value[0];
	float m_y1 = d2s.value[1];
	float m_x2 = d2s.value[3];
	float m_y2 = d2s.value[4];

	int dx = position % dst_width;
	int dy = position / dst_width;
	float src_x = m_x1 * dx + m_y1 * dy;
	float src_y = m_x2 * dx + m_y2 * dy;
	float c0, c1, c2;

	if (src_x < 0 || src_x + 1 >= src_width || src_y < 0 || src_y + 1 >= src_height) {
		c0 = const_value_st;
		c1 = const_value_st;
		c2 = const_value_st;
	}
	else {
		int x_low = floorf(src_x);
		int y_low = floorf(src_y);
		int x_high = x_low + 1;
		int y_high = y_low + 1;
		float w1 = (y_high - src_y) * (x_high - src_x);
		float w2 = (y_high - src_y) * (src_x - x_low);
		float w3 = (src_y - y_low) * (x_high - src_x);
		float w4 = (src_y - y_low) * (src_x - x_low);
		uint8_t* v1 = src + y_low * src_line_size + x_low * 3;
		uint8_t* v2 = src + y_low * src_line_size + x_high * 3;
		uint8_t* v3 = src + y_high * src_line_size + x_low * 3;
		uint8_t* v4 = src + y_high * src_line_size + x_high * 3;
		c0 = w1 * v1[0] + w2 * v2[0] + w3 * v3[0] + w4 * v4[0];
		c1 = w1 * v1[1] + w2 * v2[1] + w3 * v3[1] + w4 * v4[1];
		c2 = w1 * v1[2] + w2 * v2[2] + w3 * v3[2] + w4 * v4[2];
	}

	// bgr -> rgb
	float temp = c2;
	c2 = c0;
	c0 = temp;

	// normalization
	c0 /= 255.0f;
	c1 /= 255.0f;
	c2 /= 255.0f;
	// rgbrgbrgb -> rrrgggbbb
	int area = dst_height * dst_width;
	float* pdst_c0 = dst + dy * dst_width + dx;
	float* pdst_c1 = pdst_c0 + area;
	float* pdst_c2 = pdst_c1 + area;
	*pdst_c0 = c0;
	*pdst_c1 = c1;
	*pdst_c2 = c2;
}

void preprocess(
	uint8_t* src, const int& src_width, const int& src_height,
	float* dst, const int& dst_width, const int& dst_height,
	hipStream_t stream, float& scale) {

	AffineMatrix s2d, d2s;
	scale = std::min(dst_height / (float)src_height, dst_width / (float)src_width);
	s2d.value[0] = scale;
	s2d.value[1] = 0;
	s2d.value[2] = 0;
	s2d.value[3] = 0;
	s2d.value[4] = scale;
	s2d.value[5] = 0;
	cv::Mat m2x3_s2d(2, 3, CV_32F, s2d.value);
	cv::Mat m2x3_d2s(2, 3, CV_32F, d2s.value);
	cv::invertAffineTransform(m2x3_s2d, m2x3_d2s);

	memcpy(d2s.value, m2x3_d2s.ptr<float>(0), sizeof(d2s.value));

	int jobs = dst_height * dst_width;
	int threads = 256;
	int blocks = ceil(jobs / (float)threads);
	warpaffine_kernel << <blocks, threads, 0, stream >> > (
		src, src_width * 3, src_width,
		src_height, dst, dst_width,
		dst_height, 128, d2s, jobs);
}

__global__ void select_bbox(int kClass, int num_ancors, int keep_ancors, float conf, float* src, float* dst, int* pointer, int xyxy)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	int classid = 0;
	float maxConf = 0;
	if (idx >= num_ancors)
	{
		return;
	}
	for (int i = 4; i < kClass; ++i)
	{
		if (src[i * num_ancors + idx] >= maxConf)
		{
			maxConf = src[i * num_ancors + idx];
			classid = i - 4;
		}
	}
	if (maxConf >= conf)
	{
		if (*pointer >= keep_ancors * 6) return;
		int pointeridx = atomicAdd(pointer, 6);
		if (!xyxy)
		{
			for (int i = 0; i < 4; ++i)
			{
				dst[pointeridx + i] = src[idx + i * num_ancors];
			}
		}
		else
		{
			dst[pointeridx] = src[idx] - src[idx + 2 * num_ancors] / 2;
			dst[pointeridx + 2] = src[idx] + src[idx + 2 * num_ancors] / 2;
			dst[pointeridx + 1] = src[idx + 1 * num_ancors] - src[idx + 3 * num_ancors] / 2;
			dst[pointeridx + 3] = src[idx + 1 * num_ancors] + src[idx + 3 * num_ancors] / 2;
		}
		dst[pointeridx + 4] = src[idx + (4 + classid) * num_ancors];
		dst[pointeridx + 5] = classid;
	}
}

void postprocess(int kClass, int num_ancors, int keep_ancors, float conf, float* src, float* dst_host, int xyxy, hipStream_t stream)
{
	int point = -6;
	int* pointer;
	pointer = &point;
	int* pointer_device;
	float* dst;
	int threads = 512;
	int blocks = ceil(num_ancors / threads);
	hipMalloc((void**)&pointer_device, sizeof(int));
	hipMemcpy(pointer_device, pointer, sizeof(int), hipMemcpyHostToDevice);
	hipMalloc((void**)&dst, sizeof(float) * keep_ancors * 6);
	select_bbox << <blocks, threads, 0, stream>> > (kClass, num_ancors, keep_ancors, conf, src, dst, pointer_device, xyxy);
	hipMemcpyAsync(dst_host + 1, dst, sizeof(float) * keep_ancors * 6, hipMemcpyDeviceToHost, stream);
	hipMemcpyAsync(pointer, pointer_device, sizeof(int), hipMemcpyDeviceToHost, stream);
	*dst_host = ceil(((*pointer) + 6) / 6) + 1;
}

static float iou(float lbox[4], float rbox[4]) {
	float interBox[] = {
	  (std::max)(lbox[0] - lbox[2] / 2.f , rbox[0] - rbox[2] / 2.f), //left
	  (std::min)(lbox[0] + lbox[2] / 2.f , rbox[0] + rbox[2] / 2.f), //right
	  (std::max)(lbox[1] - lbox[3] / 2.f , rbox[1] - rbox[3] / 2.f), //top
	  (std::min)(lbox[1] + lbox[3] / 2.f , rbox[1] + rbox[3] / 2.f), //bottom
	};

	if (interBox[2] > interBox[3] || interBox[0] > interBox[1])
		return 0.0f;

	float interBoxS = (interBox[1] - interBox[0]) * (interBox[3] - interBox[2]);
	return interBoxS / (lbox[2] * lbox[3] + rbox[2] * rbox[3] - interBoxS);
}

static bool cmp(const Detection& a, const Detection& b) {
	return a.conf > b.conf;
}


void NMS(std::vector<Detection>& res, float* output, const float& conf_thresh, const float& nms_thresh) {
	int det_size = sizeof(Detection) / sizeof(float);
	std::map<float, std::vector<Detection>> m;
	for (int i = 0; i < output[0]; i++) {
		//for (int j = 0; j < 6; ++j) std::cout << output[1 + det_size * i + j] << " ";
		if (output[1 + det_size * i + 4] <= conf_thresh) continue;
		Detection det;
		memcpy(&det, &output[1 + det_size * i], det_size * sizeof(float));
		/*std::cout << det.conf << " ";*/
		if (m.count(det.class_id) == 0) m.emplace(det.class_id, std::vector<Detection>());
		m[det.class_id].push_back(det);
	}
	for (auto it = m.begin(); it != m.end(); it++) {
		auto& dets = it->second;
		std::sort(dets.begin(), dets.end(), cmp);
		for (size_t m = 0; m < dets.size(); ++m) {
			auto& item = dets[m];
			res.push_back(item);
			for (size_t n = m + 1; n < dets.size(); ++n) {
				if (iou(item.bbox, dets[n].bbox) > nms_thresh) {
					dets.erase(dets.begin() + n);
					--n;
				}
			}
		}
	}
}

void NMS_DetMsg(detect_msgs::msg::DetectMsgs& detect_msg_res, float* output, const float& conf_thresh, const float& nms_thresh) {
	int det_size = sizeof(Detection) / sizeof(float);
	std::map<float, std::vector<Detection>> m;
	for (int i = 0; i < output[0]; i++) {
		//for (int j = 0; j < 6; ++j) std::cout << output[1 + det_size * i + j] << " ";
		if (output[1 + det_size * i + 4] <= conf_thresh) continue;
		Detection det;
		memcpy(&det, &output[1 + det_size * i], det_size * sizeof(float));
		/*std::cout << det.conf << " ";*/
		if (m.count(det.class_id) == 0) m.emplace(det.class_id, std::vector<Detection>());
		m[det.class_id].push_back(det);
	}
	for (auto it = m.begin(); it != m.end(); it++) {
		auto& dets = it->second;
		std::sort(dets.begin(), dets.end(), cmp);
		for (size_t m = 0; m < dets.size(); ++m) {
			auto& item = dets[m];
			detect_msgs::msg::DetectBaseMsg item_detmsg;
			item_detmsg.bbox[0] = item.bbox[0];
			item_detmsg.bbox[1] = item.bbox[1];
			item_detmsg.bbox[2] = item.bbox[2];
			item_detmsg.bbox[3] = item.bbox[3];
			item_detmsg.class_id = item.class_id;
			item_detmsg.conf = item.conf;
			detect_msg_res.detect_msg.push_back(item_detmsg);
			for (size_t n = m + 1; n < dets.size(); ++n) {
				if (iou(item.bbox, dets[n].bbox) > nms_thresh) {
					dets.erase(dets.begin() + n);
					--n;
				}
			}
		}
	}
}

//cv::Rect getRect(cv::Mat& img, float bbox[4], float& scale) {
//	float l, r, t, b;
//	l = bbox[0] / scale;
//	t = bbox[1] / scale;
//	r = bbox[2] / scale;
//	b = bbox[3] / scale;
//	return cv::Rect(int(l), int(t), int(r - l), int(b - t));
//}
//
//void drawBbox(cv::Mat& img, std::vector<Detection>& res, float& scale, std::map<int, std::string>& Labels) {
//	for (size_t j = 0; j < res.size(); j++) {
//		cv::Rect r = getRect(img, res[j].bbox, scale);
//		std::string name = Labels[(int)res[j].class_id];
//		cv::rectangle(img, r, cv::Scalar(0xFF, 0xFF, 0), 2);
//		cv::putText(img, name, cv::Point(r.x, r.y - 1), cv::FONT_HERSHEY_PLAIN, 1.2, cv::Scalar(0xFF, 0xFF, 0), 2);
//	}
//}

//void drawBbox(cv::Mat& img, std::vector<Detection>& res, float& scale, std::map<int, std::string>& Labels) {
//	for (size_t j = 0; j < res.size(); j++) {
//		float l = res[j].bbox[0] / scale;
//		float t = res[j].bbox[1] / scale;
//		float r = res[j].bbox[2] / scale;
//		float b = res[j].bbox[3] / scale;
//		cv::Rect rect = cv::Rect(int(l), int(t), int(r - l), int(b - t));
//		std::string name = Labels[(int)res[j].class_id];
//		cv::rectangle(img, rect, cv::Scalar(0xFF, 0xFF, 0), 2);
//		cv::putText(img, name, cv::Point(rect.x, rect.y - 1), cv::FONT_HERSHEY_PLAIN, 1.2, cv::Scalar(0xFF, 0xFF, 0), 2);
//	}
//}

std::string to_string_with_precision(float num, int precision) {
	std::ostringstream oss;
	oss << std::fixed << std::setprecision(precision) << num;
	return oss.str();
}

std::vector<cv::Scalar> palette = {
	  cv::Scalar(220, 20, 60), cv::Scalar(119, 11, 32), cv::Scalar(0, 0, 142), cv::Scalar(0, 0, 230), cv::Scalar(106, 0, 228),
	  cv::Scalar(0, 60, 100), cv::Scalar(0, 80, 100), cv::Scalar(0, 0, 70), cv::Scalar(0, 0, 192), cv::Scalar(250, 170, 30),
	  cv::Scalar(100, 170, 30), cv::Scalar(220, 220, 0), cv::Scalar(175, 116, 175), cv::Scalar(250, 0, 30),
	  cv::Scalar(165, 42, 42), cv::Scalar(255, 77, 255), cv::Scalar(0, 226, 252), cv::Scalar(182, 182, 255),
	  cv::Scalar(0, 82, 0), cv::Scalar(120, 166, 157), cv::Scalar(110, 76, 0), cv::Scalar(174, 57, 255),
	  cv::Scalar(199, 100, 0), cv::Scalar(72, 0, 118), cv::Scalar(255, 179, 240), cv::Scalar(0, 125, 92),
	  cv::Scalar(209, 0, 151), cv::Scalar(188, 208, 182), cv::Scalar(0, 220, 176), cv::Scalar(255, 99, 164),
	  cv::Scalar(92, 0, 73), cv::Scalar(133, 129, 255), cv::Scalar(78, 180, 255), cv::Scalar(0, 228, 0),
	  cv::Scalar(174, 255, 243), cv::Scalar(45, 89, 255), cv::Scalar(134, 134, 103), cv::Scalar(145, 148, 174),
	  cv::Scalar(255, 208, 186), cv::Scalar(197, 226, 255), cv::Scalar(171, 134, 1), cv::Scalar(109, 63, 54),
	  cv::Scalar(207, 138, 255), cv::Scalar(151, 0, 95), cv::Scalar(9, 80, 61), cv::Scalar(84, 105, 51),
	  cv::Scalar(74, 65, 105), cv::Scalar(166, 196, 102), cv::Scalar(208, 195, 210), cv::Scalar(255, 109, 65),
	  cv::Scalar(0, 143, 149), cv::Scalar(179, 0, 194), cv::Scalar(209, 99, 106), cv::Scalar(5, 121, 0),
	  cv::Scalar(227, 255, 205), cv::Scalar(147, 186, 208), cv::Scalar(153, 69, 1), cv::Scalar(3, 95, 161),
	  cv::Scalar(163, 255, 0), cv::Scalar(119, 0, 170), cv::Scalar(0, 182, 199), cv::Scalar(0, 165, 120),
	  cv::Scalar(183, 130, 88), cv::Scalar(95, 32, 0), cv::Scalar(130, 114, 135), cv::Scalar(110, 129, 133),
	  cv::Scalar(166, 74, 118), cv::Scalar(219, 142, 185), cv::Scalar(79, 210, 114), cv::Scalar(178, 90, 62),
	  cv::Scalar(65, 70, 15), cv::Scalar(127, 167, 115), cv::Scalar(59, 105, 106), cv::Scalar(142, 108, 45),
	  cv::Scalar(196, 172, 0), cv::Scalar(95, 54, 80), cv::Scalar(128, 76, 255), cv::Scalar(201, 57, 1),
	  cv::Scalar(246, 0, 122), cv::Scalar(191, 162, 208)
};

void drawBbox(cv::Mat& img, std::vector<Detection>& res, float& scale, std::map<int, std::string>& Labels) {
	int fontFace = cv::FONT_HERSHEY_SIMPLEX;
	double fontScale = 0.7;
	int thickness = 1;
	int baseline = 0;
	for (size_t j = 0; j < res.size(); j++) {
		float l = res[j].bbox[0] / scale;
		float t = res[j].bbox[1] / scale;
		float r = res[j].bbox[2] / scale;
		float b = res[j].bbox[3] / scale;
		cv::Rect rect = cv::Rect(int(l), int(t), int(r - l), int(b - t));
		std::string name = Labels[(int)res[j].class_id] + ": ";
		std::string text = name + to_string_with_precision((float)res[j].conf, 2);
		cv::rectangle(img, rect, palette[(int)res[j].class_id], 2);
		cv::Size textSize = cv::getTextSize(text, fontFace, fontScale, thickness, &baseline);
		cv::Rect backgroundRect(cv::Point2i(rect.x, rect.y - textSize.height), textSize);
		cv::rectangle(img, backgroundRect, palette[(int)res[j].class_id], cv::FILLED);
		cv::putText(img, text, cv::Point(rect.x, rect.y - 1), fontFace, fontScale, cv::Scalar(255, 255, 255), thickness);
	}
}

void drawBbox_DetMsg(cv::Mat& img, detect_msgs::msg::DetectMsgs& detect_msg_res, float& scale, std::map<int, std::string>& Labels) {
	int fontFace = cv::FONT_HERSHEY_SIMPLEX;
	double fontScale = 0.7;
	int thickness = 1;
	int baseline = 0;
	for (size_t j = 0; j < detect_msg_res.detect_msg.size(); j++) {
		float l = detect_msg_res.detect_msg[j].bbox[0] / scale;
		float t = detect_msg_res.detect_msg[j].bbox[1] / scale;
		float r = detect_msg_res.detect_msg[j].bbox[2] / scale;
		float b = detect_msg_res.detect_msg[j].bbox[3] / scale;
		cv::Rect rect = cv::Rect(int(l), int(t), int(r - l), int(b - t));
		std::string name = Labels[(int)detect_msg_res.detect_msg[j].class_id] + ": ";
		std::string text = name + to_string_with_precision((float)detect_msg_res.detect_msg[j].conf, 2);
		cv::rectangle(img, rect, palette[(int)detect_msg_res.detect_msg[j].class_id], 2);
		cv::Size textSize = cv::getTextSize(text, fontFace, fontScale, thickness, &baseline);
		cv::Rect backgroundRect(cv::Point2i(rect.x, rect.y - textSize.height), textSize);
		cv::rectangle(img, backgroundRect, palette[(int)detect_msg_res.detect_msg[j].class_id], cv::FILLED);
		cv::putText(img, text, cv::Point(rect.x, rect.y - 1), fontFace, fontScale, cv::Scalar(255, 255, 255), thickness);
	}
}